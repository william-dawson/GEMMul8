#include "../include/gemmul8.hpp"
#include "eval.hpp"
#include "getWatt.hpp"
#include "make_matrix.hpp"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iomanip>
#include <iostream>
#include <nvml.h>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#define AVERAGE 100
#define SEED    123456
#define PHI     0.0, 0.5, 1, 1.5
// #define NUM_MODULI 6, 7, 8
#define NUM_MODULI 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15
#if defined(GPU_MEM_MB) && GPU_MEM_MB >= 21000
    #define SIZE 1024, 2048, 4096, 8192, 16384
#else
    #define SIZE 1024, 2048, 4096, 8192, 12288
#endif

#if CUBLAS_VER_MAJOR > 12 || (CUBLAS_VER_MAJOR == 12 && CUBLAS_VER_MINOR >= 9)
    #define CUBLAS_VERSION_129
#endif

#if defined(cuMpSGEMM_FLAG)
    #include "cumpsgemm/cumpsgemm.hpp"
void gemm_FP16TCEC_SCALING(cumpsgemm::handle_t const cuMpSGEMM_handle,
                           const hipblasOperation_t op_A,
                           const hipblasOperation_t op_B,
                           const unsigned m,
                           const unsigned n,
                           const unsigned k,
                           const float alpha,
                           float *const a_ptr,
                           const unsigned lda,
                           float *const b_ptr,
                           const unsigned ldb,
                           const float beta,
                           float *const c_ptr,
                           const unsigned ldc) {
    const cuMpSGEMM_compute_mode_t compute_mode = CUMPSGEMM_FP16TCEC;
    unsigned module_stage                       = 0;

    cumpsgemm::exp_stats_ext<float>(cuMpSGEMM_handle, (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), a_ptr, lda);
    unsigned exp_stats_id_A = cumpsgemm::get_current_exp_stats_buffer_id(cuMpSGEMM_handle);
    cumpsgemm::scale_A<float>(cuMpSGEMM_handle, exp_stats_id_A, 1, (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), a_ptr, lda);

    cumpsgemm::exp_stats_ext<float>(cuMpSGEMM_handle, (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), b_ptr, ldb);
    unsigned exp_stats_id_B = cumpsgemm::get_current_exp_stats_buffer_id(cuMpSGEMM_handle);
    cumpsgemm::scale_B<float>(cuMpSGEMM_handle, exp_stats_id_B, 1, (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), b_ptr, ldb);

    cumpsgemm::gemm<float>(cuMpSGEMM_handle, op_A, op_B, m, n, k, &alpha, a_ptr, lda, b_ptr, ldb, &beta, c_ptr, ldc, compute_mode, &module_stage);

    cumpsgemm::scale_C<float>(cuMpSGEMM_handle, exp_stats_id_A, exp_stats_id_B, 1, m, n, c_ptr, ldc);

    cumpsgemm::reset_scale_A<float>(cuMpSGEMM_handle, exp_stats_id_A, 1, (op_A == HIPBLAS_OP_N ? m : k), (op_A == HIPBLAS_OP_N ? k : m), a_ptr, lda);
    cumpsgemm::reset_scale_B<float>(cuMpSGEMM_handle, exp_stats_id_B, 1, (op_B == HIPBLAS_OP_N ? k : n), (op_B == HIPBLAS_OP_N ? n : k), b_ptr, ldb);
}
#endif

std::string getDeviceName() {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    std::string deviceName = deviceProp.name;

    for (char &c : deviceName) {
        if (c == ' ' || c == '/' || c == '\\') {
            c = '_';
        }
    }
    return deviceName;
}

std::string getCurrentDateTime(std::chrono::system_clock::time_point &now) {
    now                  = std::chrono::system_clock::now();
    std::time_t now_time = std::chrono::system_clock::to_time_t(now);

    std::stringstream ss;
    ss << std::put_time(std::localtime(&now_time), "%Y-%m-%d_%H-%M-%S");
    return ss.str();
}

void accuracy_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_f_accuracy_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    std::vector<float> phi_list{PHI};
    std::vector<size_t> k_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const size_t m = 1024;
    const size_t n = 1024;

    //--------------------
    // workspace
    //--------------------
    const size_t k_max          = *max_element(begin(k_list), end(k_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *workd_cpu           = new double[m * n];
    float *workf_cpu            = new float[m * n];
    size_t worksize             = gemmul8::workSize(m, n, k_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, (m * k_max + k_max * n + m * n) * (sizeof(double) + sizeof(float)));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    outFile << "phi,function,";
    std::cout << "phi,function,";
    for (auto &moduli : num_moduli_list) {
        outFile << moduli << ",";
        std::cout << moduli << ",";
    }
    outFile << std::endl;
    std::cout << std::endl;

    for (auto &phi : phi_list) {
        for (auto &k : k_list) {
            double *cpuCd = workd_cpu;
            float *cpuCf  = workf_cpu;
            double *devAd = reinterpret_cast<double *>(work_gpu);
            double *devBd = devAd + m * k;
            double *devCd = devBd + k * n;
            float *devAf  = reinterpret_cast<float *>(devCd + m * n);
            float *devBf  = devAf + m * k;
            float *devCf  = devBf + k * n;
            double errmax, errmed;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<float>(m, k, devAf, phi, seed);
            makemat::randmat<float>(k, n, devBf, phi, seed);

            //--------------------
            // C1+C2 := A*B by FP64
            //--------------------
            double alpha = 1.0;
            double beta  = 0.0;
            makemat::f2d(m, k, devAf, devAd);
            makemat::f2d(k, n, devBf, devBd);
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devAd, HIP_R_64F, m, devBd, HIP_R_64F, k, &beta, devCd, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCd, devCd, m * n * sizeof(double), hipMemcpyDeviceToHost);

            //--------------------
            // C := A*B by FP32
            //--------------------
            float alphaf = 1.0f;
            float betaf  = 0.0f;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);

            outFile << phi << ",SGEMM (k=" + std::to_string(k) + "),";
            std::cout << phi << ",SGEMM (k=" + std::to_string(k) + "),";
            for (int i = 0; i < num_moduli_list.size(); ++i) {
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            //--------------------
            // C := A*B by FP32 with HIPBLAS_COMPUTE_32F_FAST_TF32
            //--------------------
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_TF32, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);

            outFile << phi << ",SGEMM-TF32 (k=" + std::to_string(k) + "),";
            std::cout << phi << ",SGEMM-TF32 (k=" + std::to_string(k) + "),";
            for (int i = 0; i < num_moduli_list.size(); ++i) {
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            //--------------------
            // C := A*B by FP32 with CUBLAS_COMPUTE_32F_EMULATED_16BFX9
            //--------------------
#if defined(CUBLAS_VERSION_129)
            if (prop.major * 10 + prop.minor >= 100) {
                hipDeviceSynchronize();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, CUBLAS_COMPUTE_32F_EMULATED_16BFX9, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);

                outFile << phi << ",SGEMM-BF16X9 (k=" + std::to_string(k) + "),";
                std::cout << phi << ",SGEMM-BF16X9 (k=" + std::to_string(k) + "),";
                for (int i = 0; i < num_moduli_list.size(); ++i) {
                    outFile << errmax << ",";
                    std::cout << errmax << ",";
                }
                outFile << std::endl;
                std::cout << std::endl;
            }
#endif

            //--------------------
            // C := A*B by cuMpSGEMM
            //--------------------
#if defined(cuMpSGEMM_FLAG)
            cumpsgemm::handle_t cuMpSGEMM_handle;
            cumpsgemm::create(cuMpSGEMM_handle);

            hipDeviceSynchronize();
            gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);

            outFile << phi << ",FP16TCEC_SCALING (k=" + std::to_string(k) + "),";
            std::cout << phi << ",FP16TCEC_SCALING (k=" + std::to_string(k) + "),";
            for (int i = 0; i < num_moduli_list.size(); ++i) {
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            cumpsgemm::destroy(cuMpSGEMM_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            outFile << phi << ",OS2-fast (k=" + std::to_string(k) + "),";
            std::cout << phi << ",OS2-fast (k=" + std::to_string(k) + "),";
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> timestmp(4, 0);
                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            outFile << phi << ",OS2-accu (k=" + std::to_string(k) + "),";
            std::cout << phi << ",OS2-accu (k=" + std::to_string(k) + "),";
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> timestmp(4);
                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, errmax, errmed);
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;
        }
    }

    delete[] workd_cpu;
    delete[] workf_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void time_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_f_time_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;

    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "TFLOPS,"
            << "total_time [sec],"
            << "conv_32f_2_8i,"
            << "hipblasGemmEx,"
            << "conv_32i_2_8u,"
            << "inverse_scaling,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "TFLOPS,"
              << "total_time [sec],"
              << "conv_32f_2_8i,"
              << "hipblasGemmEx,"
              << "conv_32i_2_8u,"
              << "inverse_scaling,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const float phi         = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const int itermax = AVERAGE;

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *workd_cpu           = new double[n_max * n_max];
    float *workf_cpu            = new float[n_max * n_max];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 3 * sizeof(float));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m           = n;
        size_t k           = n;
        double *cpuCd      = workd_cpu;
        float *cpuCf       = workf_cpu;
        float *devAf       = reinterpret_cast<float *>(work_gpu);
        float *devBf       = devAf + m * k;
        float *devCf       = devBf + k * n;
        const size_t lda8i = ((k + 15) >> 4) << 4;
        const size_t ldb8i = lda8i;
        int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
        int8_t *B8i        = A8i + lda8i * m;
        int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
        double maxerr = 0.0, mederr = 0.0;
        double time = 0.0;
        std::chrono::system_clock::time_point start, stop;

        //--------------------
        // generate matrices
        //--------------------
        makemat::randmat<float>(m, k, devAf, phi, seed);
        makemat::randmat<float>(k, n, devBf, phi, seed);

        //--------------------
        // C1+C2 := A*B by FP64
        //--------------------
        void *workd_gpu;
        hipMalloc(&workd_gpu, (m * k + k * n + m * n) * sizeof(double));
        double *devAd = reinterpret_cast<double *>(workd_gpu);
        double *devBd = devAd + m * k;
        double *devCd = devBd + k * n;
        makemat::f2d(m, k, devAf, devAd);
        makemat::f2d(k, n, devBf, devBd);

        double alpha = 1.0;
        double beta  = 0.0;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devAd, HIP_R_64F, m, devBd, HIP_R_64F, k, &beta, devCd, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        hipMemcpy(cpuCd, devCd, m * n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(workd_gpu);

        //--------------------
        // C := A*B (int8-TC)
        //--------------------
        makemat::ones(lda8i * m + ldb8i * n, A8i);
        int32_t ialpha = 1;
        int32_t ibeta  = 0;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        outFile << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        std::cout << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        //--------------------
        // C := A*B by FP32
        //--------------------
        float alphaf = 1.0f;
        float betaf  = 0.0f;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
        outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
        std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        //--------------------
        // C := A*B by FP32 with HIPBLAS_COMPUTE_32F_FAST_TF32
        //--------------------
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_TF32, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_TF32, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
        outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
        std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        //--------------------
        // C := A*B by FP32 with CUBLAS_COMPUTE_32F_EMULATED_16BFX9
        //--------------------
#if defined(CUBLAS_VERSION_129)
        if (prop.major * 10 + prop.minor >= 100) {
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, CUBLAS_COMPUTE_32F_EMULATED_16BFX9, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, CUBLAS_COMPUTE_32F_EMULATED_16BFX9, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;
        }
#endif

        //--------------------
        // C := A*B by cuMpSGEMM
        //--------------------
#if defined(cuMpSGEMM_FLAG)
        cumpsgemm::handle_t cuMpSGEMM_handle;
        cumpsgemm::create(cuMpSGEMM_handle);
        hipDeviceSynchronize();

        gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
        outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
        std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        cumpsgemm::destroy(cuMpSGEMM_handle);
#endif

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            std::vector<double> times(4, 0);
            std::vector<double> timestmp(4, 0);

            hipDeviceSynchronize();
            timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start    = std::chrono::system_clock::now();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
            }
            time = time / itermax * 1.e-9;
            for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
        }

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            std::vector<double> times(4, 0);
            std::vector<double> timestmp(4);

            hipDeviceSynchronize();
            timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start    = std::chrono::system_clock::now();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
            }
            time = time / itermax * 1.e-9;
            for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
        }
    }

    delete[] workd_cpu;
    delete[] workf_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void watt_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_f_watt_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "watt,"
            << "GFLOPS/watt,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "watt,"
              << "GFLOPS/watt,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const float phi         = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *workd_cpu           = new double[n_max * n_max];
    float *workf_cpu            = new float[n_max * n_max];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 3 * sizeof(float));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m           = n;
        size_t k           = n;
        double *cpuCd      = workd_cpu;
        float *cpuCf       = workf_cpu;
        float *devAf       = reinterpret_cast<float *>(work_gpu);
        float *devBf       = devAf + m * k;
        float *devCf       = devBf + k * n;
        const size_t lda8i = ((k + 15) >> 4) << 4;
        const size_t ldb8i = lda8i;
        int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
        int8_t *B8i        = A8i + lda8i * m;
        int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
        double maxerr = 0.0, mederr = 0.0;

        //--------------------
        // generate matrices
        //--------------------
        makemat::randmat<float>(m, k, devAf, phi, seed);
        makemat::randmat<float>(k, n, devBf, phi, seed);

        //--------------------
        // C1+C2 := A*B by FP64
        //--------------------
        void *workd_gpu;
        hipMalloc(&workd_gpu, (m * k + k * n + m * n) * sizeof(double));
        double *devAd = reinterpret_cast<double *>(workd_gpu);
        double *devBd = devAd + m * k;
        double *devCd = devBd + k * n;
        makemat::f2d(m, k, devAf, devAd);
        makemat::f2d(k, n, devBf, devBd);

        double alpha = 1.0;
        double beta  = 0.0;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devAd, HIP_R_64F, m, devBd, HIP_R_64F, k, &beta, devCd, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        hipMemcpy(cpuCd, devCd, m * n * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(workd_gpu);

        //--------------------
        // C := A*B (int8-TC)
        //--------------------
        makemat::ones(lda8i * m + ldb8i * n, A8i);
        int32_t ialpha          = 1;
        int32_t ibeta           = 0;
        std::vector<double> res = getWatt::getWatt(
            [&]() {
                hipblasGemmEx(handle,
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_N,
                             m,
                             n,
                             lda8i,
                             &ialpha,
                             A8i,
                             HIP_R_8I,
                             lda8i,
                             B8i,
                             HIP_R_8I,
                             ldb8i,
                             &ibeta,
                             C32i,
                             HIP_R_32I,
                             m,
                             HIPBLAS_COMPUTE_32I,
                             HIPBLAS_GEMM_DEFAULT);
            },
            m,
            n,
            k);

        outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        outFile << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        std::cout << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        //--------------------
        // C := A*B by FP32
        //--------------------
        float alphaf = 1.0f;
        float betaf  = 0.0f;
        res          = getWatt::getWatt(
            [&]() {
                hipblasGemmEx(handle,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             m,
                             n,
                             k,
                             &alphaf,
                             devAf,
                             HIP_R_32F,
                             m,
                             devBf,
                             HIP_R_32F,
                             k,
                             &betaf,
                             devCf,
                             HIP_R_32F,
                             m,
                             HIPBLAS_COMPUTE_32F,
                             HIPBLAS_GEMM_DEFAULT);
            },
            m,
            n,
            k);
        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
        outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
        std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        //--------------------
        // C := A*B by FP32 with HIPBLAS_COMPUTE_32F_FAST_TF32
        //--------------------
        res = getWatt::getWatt(
            [&]() {
                hipblasGemmEx(handle,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             m,
                             n,
                             k,
                             &alphaf,
                             devAf,
                             HIP_R_32F,
                             m,
                             devBf,
                             HIP_R_32F,
                             k,
                             &betaf,
                             devCf,
                             HIP_R_32F,
                             m,
                             HIPBLAS_COMPUTE_32F_FAST_TF32,
                             HIPBLAS_GEMM_DEFAULT);
            },
            m,
            n,
            k);
        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
        outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
        std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        //--------------------
        // C := A*B by FP32 with CUBLAS_COMPUTE_32F_EMULATED_16BFX9
        //--------------------
#if defined(CUBLAS_VERSION_129)
        if (prop.major * 10 + prop.minor >= 100) {
            res = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_N,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 k,
                                 &alphaf,
                                 devAf,
                                 HIP_R_32F,
                                 m,
                                 devBf,
                                 HIP_R_32F,
                                 k,
                                 &betaf,
                                 devCf,
                                 HIP_R_32F,
                                 m,
                                 CUBLAS_COMPUTE_32F_EMULATED_16BFX9,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        }
#endif

        //--------------------
        // C := A*B by cuMpSGEMM
        //--------------------
#if defined(cuMpSGEMM_FLAG)
        cumpsgemm::handle_t cuMpSGEMM_handle;
        cumpsgemm::create(cuMpSGEMM_handle);
        hipDeviceSynchronize();

        res = getWatt::getWatt(
            [&]() {
                gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
            },
            m,
            n,
            k);

        hipDeviceSynchronize();
        hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

        outFile << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
        outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
        std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        cumpsgemm::destroy(cuMpSGEMM_handle);
#endif

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {

            hipDeviceSynchronize();
            res = getWatt::getWatt(
                [&]() {
                    gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                },
                m,
                n,
                k);

            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        }

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            hipDeviceSynchronize();

            res = getWatt::getWatt(
                [&]() {
                    gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                },
                m,
                n,
                k);

            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        }
    }

    delete[] workd_cpu;
    delete[] workf_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void time_check_rect(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_f_time-rect_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;

    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "TFLOPS,"
            << "total_time [sec],"
            << "conv_32f_2_8i,"
            << "hipblasGemmEx,"
            << "conv_32i_2_8u,"
            << "inverse_scaling,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "TFLOPS,"
              << "total_time [sec],"
              << "conv_32f_2_8i,"
              << "hipblasGemmEx,"
              << "conv_32i_2_8u,"
              << "inverse_scaling,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const float phi         = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const int itermax = AVERAGE;

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *workd_cpu           = new double[n_max * n_max];
    float *workf_cpu            = new float[n_max * n_max];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 3 * sizeof(float));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m = n;

        for (auto &k : n_list) {
            double *cpuCd      = workd_cpu;
            float *cpuCf       = workf_cpu;
            float *devAf       = reinterpret_cast<float *>(work_gpu);
            float *devBf       = devAf + m * k;
            float *devCf       = devBf + k * n;
            const size_t lda8i = ((k + 15) >> 4) << 4;
            const size_t ldb8i = lda8i;
            int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
            int8_t *B8i        = A8i + lda8i * m;
            int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
            double maxerr = 0.0, mederr = 0.0;
            double time = 0.0;
            std::chrono::system_clock::time_point start, stop;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<float>(m, k, devAf, phi, seed);
            makemat::randmat<float>(k, n, devBf, phi, seed);

            //--------------------
            // C1+C2 := A*B by FP64
            //--------------------
            void *workd_gpu;
            hipMalloc(&workd_gpu, (m * k + k * n + m * n) * sizeof(double));
            double *devAd = reinterpret_cast<double *>(workd_gpu);
            double *devBd = devAd + m * k;
            double *devCd = devBd + k * n;
            makemat::f2d(m, k, devAf, devAd);
            makemat::f2d(k, n, devBf, devBd);

            double alpha = 1.0;
            double beta  = 0.0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devAd, HIP_R_64F, m, devBd, HIP_R_64F, k, &beta, devCd, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCd, devCd, m * n * sizeof(double), hipMemcpyDeviceToHost);

            hipFree(workd_gpu);

            //--------------------
            // C := A*B (int8-TC)
            //--------------------
            makemat::ones(lda8i * m + ldb8i * n, A8i);
            int32_t ialpha = 1;
            int32_t ibeta  = 0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            outFile << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            std::cout << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            //--------------------
            // C := A*B by FP32
            //--------------------
            float alphaf = 1.0f;
            float betaf  = 0.0f;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            //--------------------
            // C := A*B by FP32 with HIPBLAS_COMPUTE_32F_FAST_TF32
            //--------------------
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_TF32, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, HIPBLAS_COMPUTE_32F_FAST_TF32, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            //--------------------
            // C := A*B by FP32 with CUBLAS_COMPUTE_32F_EMULATED_16BFX9
            //--------------------
#if defined(CUBLAS_VERSION_129)
            if (prop.major * 10 + prop.minor >= 100) {
                hipDeviceSynchronize();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, CUBLAS_COMPUTE_32F_EMULATED_16BFX9, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start = std::chrono::system_clock::now();
                    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, HIP_R_32F, m, devBf, HIP_R_32F, k, &betaf, devCf, HIP_R_32F, m, CUBLAS_COMPUTE_32F_EMULATED_16BFX9, HIPBLAS_GEMM_DEFAULT);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                }
                time = time / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << "," << "," << "," << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << "," << "," << "," << "," << std::endl;
            }
#endif

            //--------------------
            // C := A*B by cuMpSGEMM
            //--------------------
#if defined(cuMpSGEMM_FLAG)
            cumpsgemm::handle_t cuMpSGEMM_handle;
            cumpsgemm::create(cuMpSGEMM_handle);
            hipDeviceSynchronize();

            gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            cumpsgemm::destroy(cuMpSGEMM_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> times(4, 0);
                std::vector<double> timestmp(4, 0);

                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start    = std::chrono::system_clock::now();
                    timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                    for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
                }
                time = time / itermax * 1.e-9;
                for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            }

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> times(4, 0);
                std::vector<double> timestmp(4);

                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start    = std::chrono::system_clock::now();
                    timestmp = gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                    for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
                }
                time = time / itermax * 1.e-9;
                for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            }
        }
    }

    delete[] workd_cpu;
    delete[] workf_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void watt_check_rect(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_f_watt-rect_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "watt,"
            << "GFLOPS/watt,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "watt,"
              << "GFLOPS/watt,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const float phi         = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *workd_cpu           = new double[n_max * n_max];
    float *workf_cpu            = new float[n_max * n_max];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 3 * sizeof(float));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m = n;

        for (auto &k : n_list) {
            double *cpuCd      = workd_cpu;
            float *cpuCf       = workf_cpu;
            float *devAf       = reinterpret_cast<float *>(work_gpu);
            float *devBf       = devAf + m * k;
            float *devCf       = devBf + k * n;
            const size_t lda8i = ((k + 15) >> 4) << 4;
            const size_t ldb8i = lda8i;
            int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
            int8_t *B8i        = A8i + lda8i * m;
            int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
            double maxerr = 0.0, mederr = 0.0;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<float>(m, k, devAf, phi, seed);
            makemat::randmat<float>(k, n, devBf, phi, seed);

            //--------------------
            // C1+C2 := A*B by FP64
            //--------------------
            void *workd_gpu;
            hipMalloc(&workd_gpu, (m * k + k * n + m * n) * sizeof(double));
            double *devAd = reinterpret_cast<double *>(workd_gpu);
            double *devBd = devAd + m * k;
            double *devCd = devBd + k * n;
            makemat::f2d(m, k, devAf, devAd);
            makemat::f2d(k, n, devBf, devBd);

            double alpha = 1.0;
            double beta  = 0.0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devAd, HIP_R_64F, m, devBd, HIP_R_64F, k, &beta, devCd, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuCd, devCd, m * n * sizeof(double), hipMemcpyDeviceToHost);

            hipFree(workd_gpu);

            //--------------------
            // C := A*B (int8-TC)
            //--------------------
            makemat::ones(lda8i * m + ldb8i * n, A8i);
            int32_t ialpha          = 1;
            int32_t ibeta           = 0;
            std::vector<double> res = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_T,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 lda8i,
                                 &ialpha,
                                 A8i,
                                 HIP_R_8I,
                                 lda8i,
                                 B8i,
                                 HIP_R_8I,
                                 ldb8i,
                                 &ibeta,
                                 C32i,
                                 HIP_R_32I,
                                 m,
                                 HIPBLAS_COMPUTE_32I,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);

            outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            outFile << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            std::cout << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            //--------------------
            // C := A*B by FP32
            //--------------------
            float alphaf = 1.0f;
            float betaf  = 0.0f;
            res          = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_N,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 k,
                                 &alphaf,
                                 devAf,
                                 HIP_R_32F,
                                 m,
                                 devBf,
                                 HIP_R_32F,
                                 k,
                                 &betaf,
                                 devCf,
                                 HIP_R_32F,
                                 m,
                                 HIPBLAS_COMPUTE_32F,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM" << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            //--------------------
            // C := A*B by FP32 with HIPBLAS_COMPUTE_32F_FAST_TF32
            //--------------------
            res = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_N,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 k,
                                 &alphaf,
                                 devAf,
                                 HIP_R_32F,
                                 m,
                                 devBf,
                                 HIP_R_32F,
                                 k,
                                 &betaf,
                                 devCf,
                                 HIP_R_32F,
                                 m,
                                 HIPBLAS_COMPUTE_32F_FAST_TF32,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);
            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-TF32" << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            //--------------------
            // C := A*B by FP32 with CUBLAS_COMPUTE_32F_EMULATED_16BFX9
            //--------------------
#if defined(CUBLAS_VERSION_129)
            if (prop.major * 10 + prop.minor >= 100) {
                res = getWatt::getWatt(
                    [&]() {
                        hipblasGemmEx(handle,
                                     HIPBLAS_OP_N,
                                     HIPBLAS_OP_N,
                                     m,
                                     n,
                                     k,
                                     &alphaf,
                                     devAf,
                                     HIP_R_32F,
                                     m,
                                     devBf,
                                     HIP_R_32F,
                                     k,
                                     &betaf,
                                     devCf,
                                     HIP_R_32F,
                                     m,
                                     CUBLAS_COMPUTE_32F_EMULATED_16BFX9,
                                     HIPBLAS_GEMM_DEFAULT);
                    },
                    m,
                    n,
                    k);
                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "SGEMM-BF16X9" << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            }
#endif

            //--------------------
            // C := A*B by cuMpSGEMM
            //--------------------
#if defined(cuMpSGEMM_FLAG)
            cumpsgemm::handle_t cuMpSGEMM_handle;
            cumpsgemm::create(cuMpSGEMM_handle);
            hipDeviceSynchronize();

            res = getWatt::getWatt(
                [&]() {
                    gemm_FP16TCEC_SCALING(cuMpSGEMM_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphaf, devAf, m, devBf, k, betaf, devCf, m);
                },
                m,
                n,
                k);

            hipDeviceSynchronize();
            hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "FP16TCEC_SCALING" << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            cumpsgemm::destroy(cuMpSGEMM_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {

                hipDeviceSynchronize();
                res = getWatt::getWatt(
                    [&]() {
                        gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, true, work_gemm);
                    },
                    m,
                    n,
                    k);

                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            }

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                hipDeviceSynchronize();

                res = getWatt::getWatt(
                    [&]() {
                        gemmul8::gemm<float>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alphaf, devAf, m, devBf, k, &betaf, devCf, m, num_moduli, false, work_gemm);
                    },
                    m,
                    n,
                    k);

                hipDeviceSynchronize();
                hipMemcpy(cpuCf, devCf, m * n * sizeof(float), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuCf, cpuCd, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            }
        }
    }

    delete[] workd_cpu;
    delete[] workf_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

int main(int argc, char **argv) {
    std::chrono::system_clock::time_point start, stop;
    std::string deviceName = getDeviceName();
    std::string startTime  = getCurrentDateTime(start);

    bool run_accuracy   = false;
    bool run_flops      = false;
    bool run_watt       = false;
    bool run_flops_rect = false;
    bool run_watt_rect  = false;
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "accuracy") {
            run_accuracy = true;
        } else if (arg == "flops") {
            run_flops = true;
        } else if (arg == "watt") {
            run_watt = true;
        } else if (arg == "flops_rect") {
            run_flops_rect = true;
        } else if (arg == "watt_rect") {
            run_watt_rect = true;
        } else if (arg == "all") {
            run_accuracy   = true;
            run_flops      = true;
            run_watt       = true;
            run_flops_rect = true;
            run_watt_rect  = true;
        }
    }

    if (run_accuracy)
        accuracy_check(deviceName, startTime);
    if (run_flops)
        time_check(deviceName, startTime);
    if (run_watt)
        watt_check(deviceName, startTime);
    if (run_flops_rect)
        time_check_rect(deviceName, startTime);
    if (run_watt_rect)
        watt_check_rect(deviceName, startTime);

    std::string endTime = getCurrentDateTime(stop);
    auto sec            = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count() * 1.e-9;
    std::cout << std::endl;
    std::cout << "=========================" << std::endl;
    std::cout << "start        : " << startTime << std::endl;
    std::cout << "end          : " << endTime << std::endl;
    std::cout << "elapsed time : " << sec << " [sec]" << std::endl;
    std::cout << "=========================" << std::endl;
    std::cout << std::endl;

    return 0;
}
