#include "../include/gemmul8.hpp"
#include "eval.hpp"
#include "getWatt.hpp"
#include "make_matrix.hpp"
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iomanip>
#include <iostream>
#include <nvml.h>
#include <random>
#include <sstream>
#include <string>
#include <vector>

#define AVERAGE 100
#define SEED    123456
#define PHI     0.5, 1, 2, 3, 4
// #define NUM_MODULI 14, 15, 16
#define NUM_MODULI 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20
#if defined(GPU_MEM_MB) && GPU_MEM_MB >= 21000
    // more than or equal to 24GB
    #define SIZE 1024, 2048, 4096, 8192, 16384
#else
    #define SIZE 1024, 2048, 4096, 8192
#endif

#if defined(ozIMMU_EF_FLAG)
    #include "ozimmu/ozimmu.hpp"
std::vector<mtk::ozimmu::compute_mode_t> mode_list{
    mtk::ozimmu::fp64_int8_6,
    mtk::ozimmu::fp64_int8_7,
    mtk::ozimmu::fp64_int8_8,
    mtk::ozimmu::fp64_int8_9,
    mtk::ozimmu::fp64_int8_10,
    mtk::ozimmu::fp64_int8_11,
    mtk::ozimmu::fp64_int8_12};
    #define ozimmu_MIN_SPLIT 6
    #define ozimmu_MAX_SPLIT mtk::ozimmu::fp64_int8_12
#endif

std::string getDeviceName() {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    std::string deviceName = deviceProp.name;

    for (char &c : deviceName) {
        if (c == ' ' || c == '/' || c == '\\') {
            c = '_';
        }
    }
    return deviceName;
}

std::string getCurrentDateTime(std::chrono::system_clock::time_point &now) {
    now                  = std::chrono::system_clock::now();
    std::time_t now_time = std::chrono::system_clock::to_time_t(now);

    std::stringstream ss;
    ss << std::put_time(std::localtime(&now_time), "%Y-%m-%d_%H-%M-%S");
    return ss.str();
}

void accuracy_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_d_accuracy_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    std::vector<double> phi_list{PHI};
    std::vector<size_t> k_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const size_t m = 1024;
    const size_t n = 1024;

    //--------------------
    // workspace
    //--------------------
    const size_t k_max          = *max_element(begin(k_list), end(k_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *work_cpu            = new double[m * n * 3];
    size_t worksize             = gemmul8::workSize(m, n, k_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, (m * k_max + k_max * n + m * n * 2) * sizeof(double));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    outFile << "phi,function,";
    std::cout << "phi,function,";
    for (auto &moduli : num_moduli_list) {
        outFile << moduli << ",";
        std::cout << moduli << ",";
    }
    outFile << std::endl;
    std::cout << std::endl;

    for (auto &phi : phi_list) {
        for (auto &k : k_list) {
            double *cpuC  = work_cpu;
            double *cpuC1 = cpuC + m * n;
            double *cpuC2 = cpuC1 + m * n;
            double *devA  = reinterpret_cast<double *>(work_gpu);
            double *devB  = devA + m * k;
            double *devC  = devB + k * n;
            double *devC1 = devC;
            double *devC2 = devC1 + m * n;
            double errmax, errmed;
            std::vector<double> err_OS2_fast;
            std::vector<double> err_OS2_accu;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<double>(m, k, devA, phi, seed);
            makemat::randmat<double>(k, n, devB, phi, seed);

            //--------------------
            // C1+C2 := A*B (double-double arithmetic)
            //--------------------
            eval::dd_gpu::simple_gemm(m, n, k, devA, devB, devC1, devC2);
            hipDeviceSynchronize();
            hipMemcpy(cpuC1, devC1, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(cpuC2, devC2, m * n * sizeof(double), hipMemcpyDeviceToHost);

            //--------------------
            // C := A*B by FP64
            //--------------------
            double alpha = 1.0;
            double beta  = 0.0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, HIP_R_64F, m, devB, HIP_R_64F, k, &beta, devC, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, errmax, errmed);

            outFile << phi << ",DGEMM (k=" + std::to_string(k) + "),";
            std::cout << phi << ",DGEMM (k=" + std::to_string(k) + "),";
            for (int i = 0; i < num_moduli_list.size(); ++i) {
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            //--------------------
            // C := A*B by ozIMMU_EF
            //--------------------
#ifdef ozIMMU_EF_FLAG
            mtk::ozimmu::gemm_list_t fp64in_gemm;
            fp64in_gemm.push_back({mtk::ozimmu::op_n, mtk::ozimmu::op_n, m, n, k, mtk::ozimmu::real, ozimmu_MAX_SPLIT});
            mtk::ozimmu::handle_t ozimmu_handle;
            mtk::ozimmu::create(&ozimmu_handle);
            mtk::ozimmu::reallocate_working_memory(ozimmu_handle, fp64in_gemm);
            int slice = ozimmu_MIN_SPLIT;
            for (auto &mode : mode_list) {
                hipDeviceSynchronize();
                mtk::ozimmu::gemm(ozimmu_handle,
                                  mtk::ozimmu::op_n,
                                  mtk::ozimmu::op_n,
                                  m,
                                  n,
                                  k,
                                  &alpha,
                                  devA,
                                  m,
                                  devB,
                                  k,
                                  &beta,
                                  devC,
                                  m,
                                  mode,
                                  mtk::ozimmu::real);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, errmax, errmed);

                outFile << phi << ",ozIMMU_EF-" + std::to_string(slice) + "(k=" + std::to_string(k) + "),";
                std::cout << phi << ",ozIMMU_EF-" + std::to_string(slice) + "(k=" + std::to_string(k) + "),";
                for (int i = 0; i < num_moduli_list.size(); ++i) {
                    outFile << errmax << ",";
                    std::cout << errmax << ",";
                }
                outFile << std::endl;
                std::cout << std::endl;

                slice++;
            }
            mtk::ozimmu::destroy(ozimmu_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            outFile << phi << ",OS2-fast (k=" + std::to_string(k) + "),";
            std::cout << phi << ",OS2-fast (k=" + std::to_string(k) + "),";
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> timestmp(4, 0);
                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, errmax, errmed);
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            outFile << phi << ",OS2-accu (k=" + std::to_string(k) + "),";
            std::cout << phi << ",OS2-accu (k=" + std::to_string(k) + "),";
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> timestmp(4);
                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, errmax, errmed);
                outFile << errmax << ",";
                std::cout << errmax << ",";
            }
            outFile << std::endl;
            std::cout << std::endl;
        }
    }

    delete[] work_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void time_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_d_time_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "TFLOPS,"
            << "total_time [sec],"
            << "conv_64f_2_8i,"
            << "hipblasGemmEx,"
            << "conv_32i_2_8u,"
            << "inverse_scaling,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "TFLOPS,"
              << "total_time [sec],"
              << "conv_64f_2_8i,"
              << "hipblasGemmEx,"
              << "conv_32i_2_8u,"
              << "inverse_scaling,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const double phi        = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const int itermax = AVERAGE;

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *work_cpu            = new double[n_max * n_max * 3];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * sizeof(double) * ((num_moduli_max >= 5) ? 3 : 4));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m           = n;
        size_t k           = n;
        double *cpuC       = work_cpu;
        double *cpuC1      = cpuC + m * n;
        double *cpuC2      = cpuC1 + m * n;
        double *devA       = reinterpret_cast<double *>(work_gpu);
        double *devB       = devA + m * k;
        double *devC       = devB + k * n;
        double *devC1      = devC;
        double *devC2      = (num_moduli_max >= 5) ? (reinterpret_cast<double *>(work_gemm)) : (devC1 + m * n);
        const size_t lda8i = ((k + 15) >> 4) << 4;
        const size_t ldb8i = lda8i;
        int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
        int8_t *B8i        = A8i + lda8i * m;
        int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
        double maxerr = 0.0, mederr = 0.0;
        double time = 0.0;
        std::chrono::system_clock::time_point start, stop;

        //--------------------
        // generate matrices
        //--------------------
        makemat::randmat<double>(m, k, devA, phi, seed);
        makemat::randmat<double>(k, n, devB, phi, seed);

        //--------------------
        // C1+C2 := A*B (double-double arithmetic)
        //--------------------
        eval::dd_gpu::simple_gemm(m, n, k, devA, devB, devC1, devC2);
        hipDeviceSynchronize();
        hipMemcpy(cpuC1, devC1, m * n * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(cpuC2, devC2, m * n * sizeof(double), hipMemcpyDeviceToHost);

        //--------------------
        // C := A*B (int8-TC)
        //--------------------
        makemat::ones(lda8i * m + ldb8i * n, A8i);
        int32_t ialpha = 1;
        int32_t ibeta  = 0;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        outFile << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        std::cout << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        //--------------------
        // C := A*B by FP64
        //--------------------
        double alpha = 1.0;
        double beta  = 0.0;
        hipDeviceSynchronize();
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, HIP_R_64F, m, devB, HIP_R_64F, k, &beta, devC, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

        time = 0.0;
        for (int iter = 0; iter < itermax; ++iter) {
            hipDeviceSynchronize();
            start = std::chrono::system_clock::now();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, HIP_R_64F, m, devB, HIP_R_64F, k, &beta, devC, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            stop = std::chrono::system_clock::now();
            time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
        }
        time = time / itermax * 1.e-9;

        outFile << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
        outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                << "," << "," << "," << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
        std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                  << "," << "," << "," << "," << std::endl;

        //--------------------
        // C := A*B by ozIMMU_EF
        //--------------------
#ifdef ozIMMU_EF_FLAG
        mtk::ozimmu::gemm_list_t fp64in_gemm;
        fp64in_gemm.push_back({mtk::ozimmu::op_n, mtk::ozimmu::op_n, m, n, k, mtk::ozimmu::real, ozimmu_MAX_SPLIT});
        mtk::ozimmu::handle_t ozimmu_handle;
        mtk::ozimmu::create(&ozimmu_handle);
        mtk::ozimmu::reallocate_working_memory(ozimmu_handle, fp64in_gemm);
        int slice = ozimmu_MIN_SPLIT;
        for (auto &mode : mode_list) {
            hipDeviceSynchronize();
            mtk::ozimmu::gemm(ozimmu_handle,
                              mtk::ozimmu::op_n,
                              mtk::ozimmu::op_n,
                              m,
                              n,
                              k,
                              &alpha,
                              devA,
                              m,
                              devB,
                              k,
                              &beta,
                              devC,
                              m,
                              mode,
                              mtk::ozimmu::real);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                mtk::ozimmu::gemm(ozimmu_handle,
                                  mtk::ozimmu::op_n,
                                  mtk::ozimmu::op_n,
                                  m,
                                  n,
                                  k,
                                  &alpha,
                                  devA,
                                  m,
                                  devB,
                                  k,
                                  &beta,
                                  devC,
                                  m,
                                  mode,
                                  mtk::ozimmu::real);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            slice++;
        }
        mtk::ozimmu::destroy(ozimmu_handle);
#endif

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            std::vector<double> times(4, 0);
            std::vector<double> timestmp(4, 0);

            hipDeviceSynchronize();
            timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start    = std::chrono::system_clock::now();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
            }
            time = time / itermax * 1.e-9;
            for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
        }

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            std::vector<double> times(4, 0);
            std::vector<double> timestmp(4);

            hipDeviceSynchronize();
            timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start    = std::chrono::system_clock::now();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
            }
            time = time / itermax * 1.e-9;
            for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
        }
    }

    delete[] work_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void watt_check(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_d_watt_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "watt,"
            << "GFLOPS/watt,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "watt,"
              << "GFLOPS/watt,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const double phi        = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *work_cpu            = new double[n_max * n_max * 3];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 5 * sizeof(double));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m           = n;
        size_t k           = n;
        double *cpuC       = work_cpu;
        double *cpuC1      = cpuC + m * n;
        double *cpuC2      = cpuC1 + m * n;
        double *devA       = reinterpret_cast<double *>(work_gpu);
        double *devB       = devA + m * k;
        double *devC       = devB + k * n;
        double *devC1      = devC + m * n;
        double *devC2      = devC1 + m * n;
        const size_t lda8i = ((k + 15) >> 4) << 4;
        const size_t ldb8i = lda8i;
        int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
        int8_t *B8i        = A8i + lda8i * m;
        int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
        double maxerr = 0.0, mederr = 0.0;

        //--------------------
        // generate matrices
        //--------------------
        makemat::randmat<double>(m, k, devA, phi, seed);
        makemat::randmat<double>(k, n, devB, phi, seed);

        //--------------------
        // C1+C2 := A*B (double-double arithmetic)
        //--------------------
        eval::dd_gpu::simple_gemm(m, n, k, devA, devB, devC1, devC2);
        hipDeviceSynchronize();
        hipMemcpy(cpuC1, devC1, m * n * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(cpuC2, devC2, m * n * sizeof(double), hipMemcpyDeviceToHost);

        //--------------------
        // C := A*B (int8-TC)
        //--------------------
        makemat::ones(lda8i * m + ldb8i * n, A8i);
        int32_t ialpha          = 1;
        int32_t ibeta           = 0;
        std::vector<double> res = getWatt::getWatt(
            [&]() {
                hipblasGemmEx(handle,
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_N,
                             m,
                             n,
                             lda8i,
                             &ialpha,
                             A8i,
                             HIP_R_8I,
                             lda8i,
                             B8i,
                             HIP_R_8I,
                             ldb8i,
                             &ibeta,
                             C32i,
                             HIP_R_32I,
                             m,
                             HIPBLAS_COMPUTE_32I,
                             HIPBLAS_GEMM_DEFAULT);
            },
            m,
            n,
            k);

        outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        outFile << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
        std::cout << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        //--------------------
        // C := A*B by FP64
        //--------------------
        double alpha = 1.0;
        double beta  = 0.0;
        hipDeviceSynchronize();
        res = getWatt::getWatt(
            [&]() {
                hipblasGemmEx(handle,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             m,
                             n,
                             k,
                             &alpha,
                             devA,
                             HIP_R_64F,
                             m,
                             devB,
                             HIP_R_64F,
                             k,
                             &beta,
                             devC,
                             HIP_R_64F,
                             m,
                             HIPBLAS_COMPUTE_64F,
                             HIPBLAS_GEMM_DEFAULT);
            },
            m,
            n,
            k);
        hipDeviceSynchronize();
        hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

        outFile << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
        outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        std::cout << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
        std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

        //--------------------
        // C := A*B by ozIMMU_EF
        //--------------------
#ifdef ozIMMU_EF_FLAG
        mtk::ozimmu::gemm_list_t fp64in_gemm;
        fp64in_gemm.push_back({mtk::ozimmu::op_n, mtk::ozimmu::op_n, m, n, k, mtk::ozimmu::real, ozimmu_MAX_SPLIT});
        mtk::ozimmu::handle_t ozimmu_handle;
        mtk::ozimmu::create(&ozimmu_handle);
        mtk::ozimmu::reallocate_working_memory(ozimmu_handle, fp64in_gemm);
        int slice = ozimmu_MIN_SPLIT;
        for (auto &mode : mode_list) {
            hipDeviceSynchronize();
            res = getWatt::getWatt(
                [&]() {
                    mtk::ozimmu::gemm(ozimmu_handle,
                                      mtk::ozimmu::op_n,
                                      mtk::ozimmu::op_n,
                                      m,
                                      n,
                                      k,
                                      &alpha,
                                      devA,
                                      m,
                                      devB,
                                      k,
                                      &beta,
                                      devC,
                                      m,
                                      mode,
                                      mtk::ozimmu::real);
                },
                m,
                n,
                k);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            slice++;
        }
        mtk::ozimmu::destroy(ozimmu_handle);
#endif

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            hipDeviceSynchronize();

            res = getWatt::getWatt(
                [&]() {
                    gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                },
                m,
                n,
                k);

            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        }

        //--------------------
        // C := A*B by ozaki-scheme2
        //--------------------
        for (auto &num_moduli : num_moduli_list) {
            hipDeviceSynchronize();

            res = getWatt::getWatt(
                [&]() {
                    gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                },
                m,
                n,
                k);

            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
        }
    }

    delete[] work_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void time_check_rect(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_d_time-rect_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "TFLOPS,"
            << "total_time [sec],"
            << "conv_64f_2_8i,"
            << "hipblasGemmEx,"
            << "conv_32i_2_8u,"
            << "inverse_scaling,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "TFLOPS,"
              << "total_time [sec],"
              << "conv_64f_2_8i,"
              << "hipblasGemmEx,"
              << "conv_32i_2_8u,"
              << "inverse_scaling,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const double phi        = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};
    const int itermax = AVERAGE;

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *work_cpu            = new double[n_max * n_max * 3];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * sizeof(double) * ((num_moduli_max >= 5) ? 3 : 4));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m = n;

        for (auto &k : n_list) {
            double *cpuC       = work_cpu;
            double *cpuC1      = cpuC + m * n;
            double *cpuC2      = cpuC1 + m * n;
            double *devA       = reinterpret_cast<double *>(work_gpu);
            double *devB       = devA + m * k;
            double *devC       = devB + k * n;
            double *devC1      = devC;
            double *devC2      = (num_moduli_max >= 5) ? (reinterpret_cast<double *>(work_gemm)) : (devC1 + m * n);
            const size_t lda8i = ((k + 15) >> 4) << 4;
            const size_t ldb8i = lda8i;
            int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
            int8_t *B8i        = A8i + lda8i * m;
            int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
            double maxerr = 0.0, mederr = 0.0;
            double time = 0.0;
            std::chrono::system_clock::time_point start, stop;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<double>(m, k, devA, phi, seed);
            makemat::randmat<double>(k, n, devB, phi, seed);

            //--------------------
            // C1+C2 := A*B (double-double arithmetic)
            //--------------------
            eval::dd_gpu::simple_gemm(m, n, k, devA, devB, devC1, devC2);
            hipDeviceSynchronize();
            hipMemcpy(cpuC1, devC1, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(cpuC2, devC2, m * n * sizeof(double), hipMemcpyDeviceToHost);

            //--------------------
            // C := A*B (int8-TC)
            //--------------------
            makemat::ones(lda8i * m + ldb8i * n, A8i);
            int32_t ialpha = 1;
            int32_t ibeta  = 0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, lda8i, &ialpha, A8i, HIP_R_8I, lda8i, B8i, HIP_R_8I, ldb8i, &ibeta, C32i, HIP_R_32I, m, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            outFile << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            std::cout << "," << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            //--------------------
            // C := A*B by FP64
            //--------------------
            double alpha = 1.0;
            double beta  = 0.0;
            hipDeviceSynchronize();
            hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, HIP_R_64F, m, devB, HIP_R_64F, k, &beta, devC, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            time = 0.0;
            for (int iter = 0; iter < itermax; ++iter) {
                hipDeviceSynchronize();
                start = std::chrono::system_clock::now();
                hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, HIP_R_64F, m, devB, HIP_R_64F, k, &beta, devC, HIP_R_64F, m, HIPBLAS_COMPUTE_64F, HIPBLAS_GEMM_DEFAULT);
                hipDeviceSynchronize();
                stop = std::chrono::system_clock::now();
                time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
            }
            time = time / itermax * 1.e-9;

            outFile << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
            outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                    << "," << "," << "," << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
            std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                      << "," << "," << "," << "," << std::endl;

            //--------------------
            // C := A*B by ozIMMU_EF
            //--------------------
#ifdef ozIMMU_EF_FLAG
            mtk::ozimmu::gemm_list_t fp64in_gemm;
            fp64in_gemm.push_back({mtk::ozimmu::op_n, mtk::ozimmu::op_n, m, n, k, mtk::ozimmu::real, ozimmu_MAX_SPLIT});
            mtk::ozimmu::handle_t ozimmu_handle;
            mtk::ozimmu::create(&ozimmu_handle);
            mtk::ozimmu::reallocate_working_memory(ozimmu_handle, fp64in_gemm);
            int slice = ozimmu_MIN_SPLIT;
            for (auto &mode : mode_list) {
                hipDeviceSynchronize();
                mtk::ozimmu::gemm(ozimmu_handle,
                                  mtk::ozimmu::op_n,
                                  mtk::ozimmu::op_n,
                                  m,
                                  n,
                                  k,
                                  &alpha,
                                  devA,
                                  m,
                                  devB,
                                  k,
                                  &beta,
                                  devC,
                                  m,
                                  mode,
                                  mtk::ozimmu::real);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start = std::chrono::system_clock::now();
                    mtk::ozimmu::gemm(ozimmu_handle,
                                      mtk::ozimmu::op_n,
                                      mtk::ozimmu::op_n,
                                      m,
                                      n,
                                      k,
                                      &alpha,
                                      devA,
                                      m,
                                      devB,
                                      k,
                                      &beta,
                                      devC,
                                      m,
                                      mode,
                                      mtk::ozimmu::real);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                }
                time = time / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << "," << "," << "," << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << "," << "," << "," << "," << std::endl;

                slice++;
            }
            mtk::ozimmu::destroy(ozimmu_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> times(4, 0);
                std::vector<double> timestmp(4, 0);

                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start    = std::chrono::system_clock::now();
                    timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                    for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
                }
                time = time / itermax * 1.e-9;
                for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            }

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                std::vector<double> times(4, 0);
                std::vector<double> timestmp(4);

                hipDeviceSynchronize();
                timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                time = 0.0;
                for (int iter = 0; iter < itermax; ++iter) {
                    hipDeviceSynchronize();
                    start    = std::chrono::system_clock::now();
                    timestmp = gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                    hipDeviceSynchronize();
                    stop = std::chrono::system_clock::now();
                    time += std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
                    for (int j = 0; j < 4; ++j) times[j] += timestmp[j];
                }
                time = time / itermax * 1.e-9;
                for (int j = 0; j < 4; ++j) times[j] = times[j] / itermax * 1.e-9;

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                        << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << 2.0 * m * n * k / time * 1.e-12 << "," << time << ","
                          << times[0] << "," << times[1] << "," << times[2] << "," << times[3] << "," << std::endl;
            }
        }
    }

    delete[] work_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

void watt_check_rect(std::string &deviceName, std::string &dateTime) {
    std::string fileName = "oz2_results_d_watt-rect_" + deviceName + "_" + dateTime + ".csv";
    std::ofstream outFile(fileName);
    outFile << std::scientific;
    std::cout << std::scientific;
    outFile << "phi,m,n,k,"
            << "function,"
            << "relerr_max,relerr_med,"
            << "watt,"
            << "GFLOPS/watt,"
            << std::endl;
    std::cout << "phi,m,n,k,"
              << "function,"
              << "relerr_max,relerr_med,"
              << "watt,"
              << "GFLOPS/watt,"
              << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //--------------------
    // settings
    //--------------------
    unsigned long long seed = SEED;
    const double phi        = 0.5;
    std::vector<size_t> n_list{SIZE};
    std::vector<unsigned> num_moduli_list{NUM_MODULI};

    //--------------------
    // workspace
    //--------------------
    const size_t n_max          = *max_element(begin(n_list), end(n_list));
    const size_t num_moduli_max = *max_element(begin(num_moduli_list), end(num_moduli_list));
    double *work_cpu            = new double[n_max * n_max * 3];
    size_t worksize             = gemmul8::workSize(n_max, n_max, n_max, num_moduli_max);
    void *work_gpu;
    hipMalloc(&work_gpu, n_max * n_max * 5 * sizeof(double));
    hipDeviceSynchronize();
    void *work_gemm;
    hipMalloc(&work_gemm, worksize);
    hipDeviceSynchronize();

    for (auto &n : n_list) {
        size_t m = n;

        for (auto &k : n_list) {
            double *cpuC       = work_cpu;
            double *cpuC1      = cpuC + m * n;
            double *cpuC2      = cpuC1 + m * n;
            double *devA       = reinterpret_cast<double *>(work_gpu);
            double *devB       = devA + m * k;
            double *devC       = devB + k * n;
            double *devC1      = devC + m * n;
            double *devC2      = devC1 + m * n;
            const size_t lda8i = ((k + 15) >> 4) << 4;
            const size_t ldb8i = lda8i;
            int8_t *A8i        = reinterpret_cast<int8_t *>(work_gemm);
            int8_t *B8i        = A8i + lda8i * m;
            int32_t *C32i      = reinterpret_cast<int32_t *>(B8i + ldb8i * n);
            double maxerr = 0.0, mederr = 0.0;

            //--------------------
            // generate matrices
            //--------------------
            makemat::randmat<double>(m, k, devA, phi, seed);
            makemat::randmat<double>(k, n, devB, phi, seed);

            //--------------------
            // C1+C2 := A*B (double-double arithmetic)
            //--------------------
            eval::dd_gpu::simple_gemm(m, n, k, devA, devB, devC1, devC2);
            hipDeviceSynchronize();
            hipMemcpy(cpuC1, devC1, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(cpuC2, devC2, m * n * sizeof(double), hipMemcpyDeviceToHost);

            //--------------------
            // C := A*B (int8-TC)
            //--------------------
            makemat::ones(lda8i * m + ldb8i * n, A8i);
            int32_t ialpha          = 1;
            int32_t ibeta           = 0;
            std::vector<double> res = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_T,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 lda8i,
                                 &ialpha,
                                 A8i,
                                 HIP_R_8I,
                                 lda8i,
                                 B8i,
                                 HIP_R_8I,
                                 ldb8i,
                                 &ibeta,
                                 C32i,
                                 HIP_R_32I,
                                 m,
                                 HIPBLAS_COMPUTE_32I,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);

            outFile << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            outFile << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "INT8-GEMM" << ",";
            std::cout << "," << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            //--------------------
            // C := A*B by FP64
            //--------------------
            double alpha = 1.0;
            double beta  = 0.0;
            hipDeviceSynchronize();
            res = getWatt::getWatt(
                [&]() {
                    hipblasGemmEx(handle,
                                 HIPBLAS_OP_N,
                                 HIPBLAS_OP_N,
                                 m,
                                 n,
                                 k,
                                 &alpha,
                                 devA,
                                 HIP_R_64F,
                                 m,
                                 devB,
                                 HIP_R_64F,
                                 k,
                                 &beta,
                                 devC,
                                 HIP_R_64F,
                                 m,
                                 HIPBLAS_COMPUTE_64F,
                                 HIPBLAS_GEMM_DEFAULT);
                },
                m,
                n,
                k);
            hipDeviceSynchronize();
            hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

            outFile << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
            outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            std::cout << phi << "," << m << "," << n << "," << k << "," << "DGEMM" << ",";
            std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

            //--------------------
            // C := A*B by ozIMMU_EF
            //--------------------
#ifdef ozIMMU_EF_FLAG
            mtk::ozimmu::gemm_list_t fp64in_gemm;
            fp64in_gemm.push_back({mtk::ozimmu::op_n, mtk::ozimmu::op_n, m, n, k, mtk::ozimmu::real, ozimmu_MAX_SPLIT});
            mtk::ozimmu::handle_t ozimmu_handle;
            mtk::ozimmu::create(&ozimmu_handle);
            mtk::ozimmu::reallocate_working_memory(ozimmu_handle, fp64in_gemm);
            int slice = ozimmu_MIN_SPLIT;
            for (auto &mode : mode_list) {
                hipDeviceSynchronize();
                res = getWatt::getWatt(
                    [&]() {
                        mtk::ozimmu::gemm(ozimmu_handle,
                                          mtk::ozimmu::op_n,
                                          mtk::ozimmu::op_n,
                                          m,
                                          n,
                                          k,
                                          &alpha,
                                          devA,
                                          m,
                                          devB,
                                          k,
                                          &beta,
                                          devC,
                                          m,
                                          mode,
                                          mtk::ozimmu::real);
                    },
                    m,
                    n,
                    k);
                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "ozIMMU_EF-" << slice << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;

                slice++;
            }
            mtk::ozimmu::destroy(ozimmu_handle);
#endif

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                hipDeviceSynchronize();

                res = getWatt::getWatt(
                    [&]() {
                        gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, true, work_gemm);
                    },
                    m,
                    n,
                    k);

                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-fast-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            }

            //--------------------
            // C := A*B by ozaki-scheme2
            //--------------------
            for (auto &num_moduli : num_moduli_list) {
                hipDeviceSynchronize();

                res = getWatt::getWatt(
                    [&]() {
                        gemmul8::gemm<double>(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m, devB, k, &beta, devC, m, num_moduli, false, work_gemm);
                    },
                    m,
                    n,
                    k);

                hipDeviceSynchronize();
                hipMemcpy(cpuC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                eval::err::gemm_err(m, n, cpuC, cpuC1, cpuC2, maxerr, mederr);

                outFile << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                outFile << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
                std::cout << phi << "," << m << "," << n << "," << k << "," << "OS2-accu-" << num_moduli << ",";
                std::cout << maxerr << "," << mederr << "," << res[0] << "," << res[1] * 1.e-9 << "," << std::endl;
            }
        }
    }

    delete[] work_cpu;
    hipFree(work_gpu);
    hipFree(work_gemm);
    hipblasDestroy(handle);
    outFile.close();
}

int main(int argc, char **argv) {
    std::chrono::system_clock::time_point start, stop;
    std::string deviceName = getDeviceName();
    std::string startTime  = getCurrentDateTime(start);

    bool run_accuracy   = false;
    bool run_flops      = false;
    bool run_watt       = false;
    bool run_flops_rect = false;
    bool run_watt_rect  = false;
    for (int i = 1; i < argc; ++i) {
        std::string arg = argv[i];
        if (arg == "accuracy") {
            run_accuracy = true;
        } else if (arg == "flops") {
            run_flops = true;
        } else if (arg == "watt") {
            run_watt = true;
        } else if (arg == "flops_rect") {
            run_flops_rect = true;
        } else if (arg == "watt_rect") {
            run_watt_rect = true;
        } else if (arg == "all") {
            run_accuracy   = true;
            run_flops      = true;
            run_watt       = true;
            run_flops_rect = true;
            run_watt_rect  = true;
        }
    }

    if (run_accuracy)
        accuracy_check(deviceName, startTime);
    if (run_flops)
        time_check(deviceName, startTime);
    if (run_watt)
        watt_check(deviceName, startTime);
    if (run_flops_rect)
        time_check_rect(deviceName, startTime);
    if (run_watt_rect)
        watt_check_rect(deviceName, startTime);

    std::string endTime = getCurrentDateTime(stop);
    auto sec            = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count() * 1.e-9;
    std::cout << std::endl;
    std::cout << "=========================" << std::endl;
    std::cout << "start        : " << startTime << std::endl;
    std::cout << "end          : " << endTime << std::endl;
    std::cout << "elapsed time : " << sec << " [sec]" << std::endl;
    std::cout << "=========================" << std::endl;
    std::cout << std::endl;

    return 0;
}
